#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "pipeline.h"

__global__ void kernel(int *x, int *y) {
    printf("%d %d\n", *x, *y);
}

int main() {
    hipDeviceReset();

    int x = 5;
    int *d_x;
    hipMalloc((void**)&d_x, sizeof(x));
    hipMemcpy(d_x, &x, sizeof(x), hipMemcpyHostToDevice);

    int y = 6;
    int *d_y;
    hipMalloc((void**)&d_y, sizeof(y));
    hipMemcpy(d_y, &y, sizeof(y), hipMemcpyHostToDevice);

    printf("Host: %p %p\n", d_x, d_y);
    
    Stage stage((void*)kernel, 2);
    stage.launch(1, 1, (void*)d_x, (void*)d_y);
    
    auto err = hipGetLastError();
    if (err) {
        puts(hipGetErrorString(err));
    }

    hipFree(d_x);
    hipFree(d_y);

    return 0;
}