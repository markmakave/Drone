#include "hip/hip_runtime.h"
#include <climits>

#include "kernels.cuh"
#include "map.h"

using lm::map;

__global__ void lm::autopilot::disparity(const map<uint8_t>* left,
                                         const map<uint8_t>* right,
                                         map<int>* disparity,
                                         const int block_radius,
                                         const int distinction_threshold,
                                         const int validation_threshold)
{
    // Current pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Out of bounds check
    if (x < block_radius || 
        y < block_radius || 
        x >= disparity->width() - block_radius || 
        y >= disparity->height() - block_radius)
    {
        disparity->at(x, y) = -1;
        return;
    }

    // Initial infimum values
    int infimum_position = x;
    int infimum_value = INT_MAX;

    // Right frame epipolar line walkthrough
    for (int center = block_radius; center <= x; ++center) {
        int hamming_sum = 0;

        // Block walkthrough
        for (int x_offset = -block_radius; x_offset <= block_radius; ++x_offset) {
            for (int y_offset = -block_radius; y_offset <= block_radius; ++y_offset) {
                int cur_difference = (int)(*left)(x + x_offset, y + y_offset) - (int)(*right)(center + x_offset, y + y_offset);
                if (abs(cur_difference) > distinction_threshold)
                    hamming_sum++;
            }
        }

        // Selecting miminum hamming sum
        if (hamming_sum < infimum_value) {
            infimum_position = center;
            infimum_value = hamming_sum;
        }
    }

    // Validation
    if (infimum_value <= validation_threshold) {
        disparity->operator()(x, y) = x - infimum_position;
    } else {
        disparity->operator()(x, y) = -1;
    }
}

__global__ void lm::autopilot::depth(const map<int>* disparity,
                                     map<float>* depth,
                                     const float focal_lenght,
                                     const float camera_distance)
{
    // Current pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Out of bounds check
    if (x >= depth->width()|| 
        y >= depth->height())
    {
        return;
    }

    // Validation
    int current = disparity->operator()(x, y);
    if (current <= 0) {
        depth->operator()(x, y) = -1.f;
    } else {
        depth->operator()(x, y) = focal_lenght * camera_distance / current;
    }
}